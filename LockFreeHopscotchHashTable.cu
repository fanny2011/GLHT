#include "hip/hip_runtime.h"
//#include"cutil.h"			// Comment this if cutil.h is not available
#include"hip/hip_runtime.h"
#include"stdio.h"

// Number of operations
//#define NUM_ITEMS 50000
// Number operations per block
#define FACTOR 1
// Number of integer keys assumed in the range [10, 9+KEYS]
//#define KEYS 100
// Number of threads per block
#define THREADS_NUM 32
// Number of hash table buckets
#define BUCKETS_NUM 36419

// Supported operations
#define ADD (0)
#define DELETE (1)
#define SEARCH (2)

#if __WORDSIZE == 64
typedef unsigned long long int LL;
#else
typedef unsigned int LL;
#endif 

// Definition of generic slot
typedef LL Slot; 

#if __WORDSIZE == 64
	// Size of the neighborhood, every bucket has (1 + NEIGHBORHOOD_SIZE) slots
	#define NEIGHBORHOOD_SIZE 31

	// Because of the flag MASK, the key value in the Slot need to be restricted
	#define MAX_KEY ((LL)0x000000000fffffff)

	// Use MASK to get the flag value in Slot
	#define EMP_FLAG_MASK ((LL)0x8000000000000000)
	#define CHECK_1_FLAG_MASK ((LL)0x4000000000000000)
	#define CHECK_2_FLAG_MASK ((LL)0x2000000000000000)
	#define SWAP_FLAG_MASK ((LL)0x1000000000000000)

	#define BITMAP_MASK ((LL)0x0ffffffff0000000)
	#define BITMAP_SHIFT 28

	#define WRONG_POS ((LL)0xffffffffffffffff)

#else 
	#define NEIGHBORHOOD_SIZE 15

	#define MAX_KEY ((LL)0x00000fff)

	#define EMP_FLAG_MASK ((LL)0x80000000)
	#define CHECK_1_FLAG_MASK ((LL)0x40000000)
	#define CHECK_2_FLAG_MASK ((LL)0x20000000)
	#define SWAP_FLAG_MASK ((LL)0x10000000)

	#define BITMAP_MASK ((LL)0x0ffff000)
	#define BITMAP_SHIFT 12

	#define WRONG_POS ((LL)0xffffffff)

#endif 

#define BIT ((LL)0x1)

#define BUCKET_RANGE (NEIGHBORHOOD_SIZE+1)
// Actuall hash table pysical size
#define TABLE_SIZE (BUCKETS_NUM + NEIGHBORHOOD_SIZE)
#define MAX_PROBES_FOR_EMPTY_BUCKET (12*BUCKET_RANGE)

__device__ Slot * m_slots;			// Array of hash table slots

// Kernel for initializing device memory
// This kernel initializes every slot as an empty node
__global__ void init(Slot * slots)
{       
  	m_slots = slots;
}

// Hash function
__device__ int Hash(LL x)
{
  	return x%BUCKETS_NUM;
}

__device__ bool CompareAndSet(int pos, LL old_value, LL new_value)
{	
	Slot old_value_out = atomicCAS(&(m_slots[pos]), old_value, new_value);
	if (old_value_out == old_value) return true;
	return false;
}

__device__ void Find(LL key, LL * result, Slot * location)
{
	int tid = threadIdx.x;
	int pos = Hash(key); // step 0
	LL bitmap;
	Slot location_pos;

	do{
		*location = m_slots[pos + tid];
		location_pos = __shfl(*location, 0);
	} while( (location_pos & CHECK_1_FLAG_MASK) != 0 ); // step 2a

	// step 2b
	if( (location_pos & EMP_FLAG_MASK) != 0 ){ // step 2b1
		bitmap = (BITMAP_MASK >> BITMAP_SHIFT);
	} else { // step 2b2
		bitmap = ( (location_pos & BITMAP_MASK) >> BITMAP_SHIFT);
	}

	int predict = 0;
	int tmp_pos = Hash((*location) & MAX_KEY);

	if( (((bitmap >> tid) & BIT) != 0) // is valid
		&& ( ( (*location) & EMP_FLAG_MASK) == 0) // no emp flag
		&& ( ( (*location) & MAX_KEY) == key ) // is the key
		&& (tmp_pos == pos) // just for safe
	){
		predict = 1;
	}

	int ans = __ffs(__ballot(predict));

	if(ans==0){
		*result = WRONG_POS;
	} else {
		*result = pos + (ans - 1);
	}
}

__device__ void Delete(LL key, LL * result)
{
	int tid = threadIdx.x;
	int pos = Hash(key); // step 0
	LL target;
	Slot location;
	Slot location_pos;
	Slot new_location_pos;
	LL ans;
	bool success;

	while (true) {
		ans = WRONG_POS;
		target = WRONG_POS;
		success = false;

		Find(key, &target, &location); // step 1

		if(target == WRONG_POS){
			*result = 0; //return false
			return; //step 2b
		}

		location_pos = __shfl(location, 0);

		if( ((location_pos & CHECK_1_FLAG_MASK) != 0)
		|| ((location_pos & CHECK_2_FLAG_MASK) != 0)
		|| ((location_pos & SWAP_FLAG_MASK) != 0) ){
			;

		} else if( ((location_pos & EMP_FLAG_MASK) == 0) 
				&& ( ( ( (location_pos & BITMAP_MASK) >> BITMAP_SHIFT) & BIT ) != 0 )
				&& ( ( location_pos & MAX_KEY) == key ) ) {

			if(tid == 0){
				new_location_pos = (location_pos | EMP_FLAG_MASK);

				success = CompareAndSet(pos, location_pos, new_location_pos);
      			if (success) {
      				ans = 1; // return true;
      			} 
			}
			
			ans = __shfl(ans, 0);
			if(ans == 1){
				*result = 1;
				return;
			}

		} else {
			new_location_pos = (location_pos | CHECK_1_FLAG_MASK);

			if(tid == 0){
				/*
				if(pos == 7468){
					printf("Delete key: %lu, step 3c add CHECK_1_FLAG_MASK\n", key);
				}*/

				success = CompareAndSet(pos, location_pos, new_location_pos);
			}
				
			success = __shfl(success, 0);
			if(success){
				location_pos = new_location_pos;
				      		
      			int lane_id = (int)target - pos;
      			
      			if(tid == lane_id){ 
      				Slot new_location = (location | EMP_FLAG_MASK);
      				success = CompareAndSet(target, location, new_location); 

      				if(success){ // step 4a
      					new_location_pos = (location_pos  & (~CHECK_1_FLAG_MASK));
      					//remove bitmap bit
      					new_location_pos &= (~(BIT<<(BITMAP_SHIFT+lane_id)));

      					success = CompareAndSet(pos, location_pos, new_location_pos); 
      					if(success){
      						ans = 1;
      					} else {
      						// TODO: design fail
      						printf("Delete key: %lu, step4a2 design fail\n", key);
      					}
      				
      				} else { //step 4b
      					new_location_pos = (location_pos & (~CHECK_1_FLAG_MASK));

      					success = CompareAndSet(pos, location_pos, new_location_pos); 
      				
      					if (!success) {
      						// TODO: design fail
      						printf("Delete key: %lu, step4b2 design fail\n", key);
      					} 
      				}

      			}

      			ans = __shfl(ans, lane_id);
				if(ans == 1){
					*result = 1;
					return;
				}

      		} // else step 3c2

		}

	} 

}

__device__ void Insert(LL key, LL * result)
{
	int tid = threadIdx.x;
	int pos = Hash(key); // step 0
	LL target;
	Slot location;
	Slot location_pos;
	Slot new_location_pos;
	LL ans;
	bool success;
	//Slot location_swap_empty;
	Slot location_swap;
	Slot location_check2;
	int search_pos = pos;

	while (true) {
		ans = WRONG_POS;
		target = WRONG_POS;
		success = false;

		Find(key, &target, &location); // step 1

		if(target != WRONG_POS){
			*result = 0; // return false
			return; // step 2b
		}

		location_pos = __shfl(location, 0);

		// step 3
		if( ((location_pos & CHECK_1_FLAG_MASK) != 0)
		|| ((location_pos & CHECK_2_FLAG_MASK) != 0)
		|| ((location_pos & SWAP_FLAG_MASK) != 0) ){ // step 3a
			continue;
		} else if( (location_pos & EMP_FLAG_MASK) != 0  ){ // step 3b

			if(tid == 0){
				new_location_pos = (key & MAX_KEY);
				// add bitmap bit;
				new_location_pos |= (BIT<<(BITMAP_SHIFT));
				success = CompareAndSet(pos, location_pos, new_location_pos);

				if(success){
					ans = 1;
				}
			}
			
			ans = __shfl(ans, 0);
			if(ans == 1){ // step 3b1
				*result = 1;
				return;
			} else { // step 3b2
				continue;
			}

		} else { // step 3c
			bool continue_3c = false;

			if(tid == 0){
				new_location_pos = (location_pos | CHECK_1_FLAG_MASK);
				/*
				if(pos == 7468){
					printf("Insert key: %lu, step 3c add CHECK_1_FLAG_MASK\n", key);
					printf("location_pos: %x%x, new_location_pos: %x%x\n", location_pos, new_location_pos);
				}*/

				success = CompareAndSet(pos, location_pos, new_location_pos);

				if(!success){ // step 3c2		
					continue_3c = true;
				} else {
					location = new_location_pos;
				}
			}

			continue_3c = __shfl(continue_3c, 0);
			if(continue_3c) continue;

			location_pos = __shfl(location, 0);			
		}

		search_pos = pos;
step_4:
		//__syncthreads();

		bool condition_4b = (((location & CHECK_1_FLAG_MASK) != 0) && ((location & EMP_FLAG_MASK) != 0));
		LL target_4b = __ffs(__ballot(condition_4b));

		bool condition_4a = (((location & CHECK_1_FLAG_MASK) == 0) && ((location & EMP_FLAG_MASK) != 0));
		LL target_4a_list = __ballot(condition_4a);

		for(int target_4a_offset =  __ffs(target_4a_list); 
				target_4a_offset != 0; 
				target_4a_offset = __ffs(target_4a_list) ){

			LL lanid_4a = target_4a_offset-1;

			target = search_pos + lanid_4a;
			
			bool goto_4a_step7 = false;
			if(tid == lanid_4a){
				Slot new_location = (location | CHECK_1_FLAG_MASK);
				/*
				if(target == 7468){
					printf(" Delete key: %lu, step 4a add CHECK_1_FLAG_MASK\n", key);
				}*/

				success = CompareAndSet(target, location, new_location);
				if(success){
					location = new_location;
					//location_swap_empty = location;
					goto_4a_step7 = true;
				}
			}

			goto_4a_step7 = __shfl(goto_4a_step7, lanid_4a);

			if(goto_4a_step7){
				//target = __shfl(target, lanid_4a);
				//location_swap_empty = __shfl(location_swap_empty, lanid_4a); // use for swap
				goto step_7;  
			}

			// bug fixed: should be target_4a_list &= (~(BIT<<lanid_4a));
			// target_4a_list &= (~(BIT<<target_4a_list));
			target_4a_list &= (~(BIT<<lanid_4a));
		}

		if(target_4b != 0){
			search_pos = pos;
			goto step_6;
		}

		// step 5
		search_pos += BUCKET_RANGE;

		if( search_pos >= pos + MAX_PROBES_FOR_EMPTY_BUCKET || search_pos >= BUCKETS_NUM ){
			bool goto_5a_full = false;

			if(tid == 0){
				new_location_pos = (location_pos & (~CHECK_1_FLAG_MASK));

				success = CompareAndSet(pos, location_pos, new_location_pos);
				if(success){ // step 5a1
					goto_5a_full = true;
				} else { // step 5a2
					// TODO: design fail
					printf("Insert key: %lu, step5a2 design fail\n", key);
				}

			}
			
			goto_5a_full = __shfl(goto_5a_full, 0);
			if(goto_5a_full){
				// TODO: full
				return;
			} else {
				// TODO: design fail 5a2
				printf("Insert key: %lu, step5a2 full design fail\n", key);
			}
		}

step_6:
		location = m_slots[search_pos + tid];
		goto step_4;

step_7:
		if( ((int)target > pos) && (((int)target - NEIGHBORHOOD_SIZE) <= pos) ){ // step 7a
			location = m_slots[pos + tid];
			location_pos = __shfl(location,0);

			int lanid_7a = (int)target - pos;

			if(tid == lanid_7a){
				Slot new_location_target = (key & MAX_KEY);
				success = CompareAndSet(target, location, new_location_target);
				if(success){
					new_location_pos = (location_pos & (~CHECK_1_FLAG_MASK));
					//add bitmap bit
					new_location_pos |= (BIT<<(BITMAP_SHIFT+lanid_7a));
					/*
					if(pos == 7468){
						printf("Insert key: %lu, step 7a remove CHECK_1_FLAG_MASK\n", key);
						printf("location_pos: %x%x, new_location_pos: %x%x\n", location_pos, new_location_pos);
					}*/

					success = CompareAndSet(pos, location_pos, new_location_pos);
					if(success){
						ans = 1;
					} else {
						// TODO: design fail
						printf("Insert key: %lu, step7a1b design fail\n", key);
					}

				} else {
					new_location_pos = (location_pos & (~CHECK_1_FLAG_MASK));
					success = CompareAndSet(pos, location_pos, new_location_pos);
					if( !success ){
						// TODO: design fail
						printf("Insert key: %lu, step7a2b design fail\n", key);
					}
				}
			}

			ans = __shfl(ans, lanid_7a);
			if(ans == 1){ // step 3a1a
				*result = 1;
				return;
			} else {
				continue; 
			}
		}

		//step 8
		int to_check_2 = target - NEIGHBORHOOD_SIZE;
		location = m_slots[to_check_2 + tid];
		location_check2 = __shfl(location, 0);
step_9:
		if( ((location_check2 & CHECK_1_FLAG_MASK) == 0)
			&& ((location_check2 & CHECK_2_FLAG_MASK) == 0)
			&& ((location_check2 & SWAP_FLAG_MASK) == 0) 
			&& ((location_check2 & EMP_FLAG_MASK) == 0) ){ // step 9a

			bool goto_9a1_step12 = false;
			if(tid == 0){
				Slot new_location_check2 = (location_check2 | CHECK_2_FLAG_MASK);
				new_location_check2 |= (BIT<<(BITMAP_SHIFT+(target-to_check_2)));

				success = CompareAndSet(to_check_2, location_check2, new_location_check2);
				if(success){
					location = new_location_check2;
					location_check2 = location;
					goto_9a1_step12 = true;
				}
			} 

			goto_9a1_step12 = __shfl(goto_9a1_step12, 0);
			if(goto_9a1_step12){
				location_check2 = __shfl(location_check2, 0);
				goto step_12;
			}

		} else if( ((location_check2 & CHECK_1_FLAG_MASK) == 0)
			&& ((location_check2 & EMP_FLAG_MASK) != 0) ) { // step 9b
			// bug fixed: add CHECK_1_FLAG_MASK to location_check2 and remove target’s CHECK_1_FLAG_MASK

			int lanid_9b = (int)target - to_check_2;
			bool goto_9b_step7 = false;

			if(tid == lanid_9b){
				Slot new_location_check2 = (location_check2 | CHECK_1_FLAG_MASK);

				success = CompareAndSet(to_check_2, location_check2, new_location_check2);
				if(success){
					location_check2 = new_location_check2;

					Slot new_location_target = (location & (~CHECK_1_FLAG_MASK));

					success = CompareAndSet(target, location, new_location_target);
					if(success){
						goto_9b_step7 = true;
					} else {
						// TODO : design fail
					}
				} else {
					// TODO : design fail
				}
			}
			
			location_check2 = __shfl(location_check2,lanid_9b);
			goto_9b_step7 = __shfl(goto_9b_step7,lanid_9b);

			if(goto_9b_step7){
				target = to_check_2;
				goto step_7;
			}
			
		} else if( ((location_check2 & CHECK_1_FLAG_MASK) != 0)
			&& ((location_check2 & EMP_FLAG_MASK) != 0) ) { // step 9c
			// bug fixed: remove target’s CHECK_1_FLAG_MASK and change search_pos = to_check_2

			int lanid_9c = (int)target - to_check_2;
			bool goto_9c_step6 = false;

			if(tid == lanid_9c){
				Slot new_location_target = (location & (~CHECK_1_FLAG_MASK));

				success = CompareAndSet(target, location, new_location_target);
				if(success){
					goto_9c_step6 = true;
				} else {
					// TODO : design fail
				}
			}
			
			goto_9c_step6 = __shfl(goto_9c_step6,lanid_9c);

			if(goto_9c_step6){
				search_pos = to_check_2;
				goto step_6;
			}
		}

step_10:
		to_check_2++;
		location = m_slots[to_check_2 + tid];

		//step 11
		if(to_check_2 < (int)target){
			location_check2 = __shfl(location, 0);
			goto step_9;
		} else { // to_check_2 == target

			bool goto_11b1_full = false;

			if(tid == 0){
				Slot new_location = (location & (~CHECK_1_FLAG_MASK));
				success = CompareAndSet(target, location, new_location);
				if(success){
					location = new_location;
					
					new_location_pos = (location_pos & (~CHECK_1_FLAG_MASK));
					success = CompareAndSet(pos, location_pos, new_location_pos);
					if(success){ // step 11b1a
						goto_11b1_full = true;
					} else { // step 11b1b
						// TODO: design fail
						printf("Insert key: %lu, step11b1b design fail\n", key);
					}

				} else { // step 11b2
					// TODO: design fail
					printf("Insert key: %lu, step11b2 design fail\n", key);
				}
			}
			
			goto_11b1_full = __shfl(goto_11b1_full, 0);
			if(goto_11b1_full){
				// TODO: full
				return;

			} else {
				// TODO: design fail 11b2
				printf("Insert key: %lu, step11b2 full design fail\n", key);
			}

		}

step_12:
		if ( (location_check2 & (BIT<<BITMAP_SHIFT)) != 0 ){ // step 12a;

			int lanid_12a = target - to_check_2;

			bool goto_12a1a_step7 = false;

			if(tid == lanid_12a){
				Slot new_location = (location & (~MAX_KEY)) | (location_check2 & MAX_KEY);
				new_location &= (~EMP_FLAG_MASK);
				new_location &= (~CHECK_1_FLAG_MASK);
				new_location &= (~BITMAP_MASK);

				success = CompareAndSet(target, location, new_location);

				if(success){ // step 12a1
					location = new_location;

					Slot new_location_check2 = (location_check2 & (~CHECK_2_FLAG_MASK)) | EMP_FLAG_MASK;
					success = CompareAndSet(to_check_2, location_check2, new_location_check2);

					if(success){ // step 12a1a
						location_check2 = new_location_check2;
						goto_12a1a_step7 = true;
					} else { // step 12a1b
						// TODO: design fail
						printf("Insert key: %lu, step12a1b design fail\n", key);
					}

				} else { // step 12a2
					// TODO: design fail
					printf("Insert key: %lu, step12a2 design fail\n", key);
				}
			}

			goto_12a1a_step7 = __shfl(goto_12a1a_step7, lanid_12a);
			if(goto_12a1a_step7){
				location_check2 = __shfl(location_check2, lanid_12a);
				// bug fixed: target = to_check_2
				target = to_check_2;
				goto step_7;
			}

		} 

		// step 12b   

		LL bitmap = ( (location_check2 & BITMAP_MASK) >> BITMAP_SHIFT);
		int predict = 0; 

		if( (((bitmap >> tid) & BIT) != 0) // is valid
			&& ((location & CHECK_1_FLAG_MASK) == 0)
			&& ((location & CHECK_2_FLAG_MASK) == 0)
			&& ((location & SWAP_FLAG_MASK) == 0) ){
			predict = 1;
		}

		LL swap_list = __ballot(predict);

		// step 13
		for(int to_swap_offset =  __ffs(swap_list); 
				to_swap_offset != 0 && (to_swap_offset-1) < (int)target-to_check_2 ; 
				to_swap_offset = __ffs(swap_list) ){

			to_swap_offset--;

			// step 14
			int to_swap = to_check_2 + to_swap_offset;
			location_swap = __shfl(location, to_swap_offset);
			int lanid_target = target-to_check_2;

			// TODO: lanid == to_swap_offset 's location need to change?

			if( (location_swap & EMP_FLAG_MASK) != 0 ){ // step 14a

				bool goto_14a1a1_step7 = false;
				bool goto_14a1a2_step6 = false;

				// bug fixed: first put CHECK_1_FLAG_MASK on location_swap 

				if(tid == lanid_target){

					Slot new_location_swap = location_swap | CHECK_1_FLAG_MASK;
					success = CompareAndSet(to_swap, location_swap, new_location_swap);
					if(success){ // step 14a1
						//location_swap = new_location_swap;

						Slot new_location_check2 = (location_check2 & (~CHECK_2_FLAG_MASK));
						new_location_check2 &= (~(BIT<<(BITMAP_SHIFT+(lanid_target))));

						success = CompareAndSet(to_check_2, location_check2, new_location_check2);
						if(success){ // step 14a1a
							location_check2 = new_location_check2;

							Slot new_location_target = (location & (~CHECK_1_FLAG_MASK));

							success = CompareAndSet(target, location, new_location_target);
							if(success){ // step 14a1a1
								location = new_location_target;
								
								target = to_swap;
								goto_14a1a1_step7 = true;

							} else { // step 14a1a2
							  	// bug fixed: change search_pos = to_check_2;
								//search_pos = pos;
								search_pos = to_check_2;
								goto_14a1a2_step6 = true;

							}

						} else { // step 14a1b
							// TODO: design fail
							printf("Insert key: %lu, step14a1b design fail\n", key);
						}

					} else { // step 14a2
						// TODO: design fail
						printf("Insert key: %lu, step14a2 design fail\n", key);
					}

				}

				location_check2 = __shfl(location_check2, lanid_target);
				goto_14a1a1_step7 = __shfl(goto_14a1a1_step7, lanid_target);
				goto_14a1a2_step6 = __shfl(goto_14a1a2_step6, lanid_target);

				if(goto_14a1a1_step7){
					target = __shfl(target, lanid_target);
					goto step_7;
				}

				if(goto_14a1a2_step6){
					search_pos = __shfl(search_pos, lanid_target);
					goto step_6;
				}

			} else { // step 14b

				bool goto_14b1a1a_step7 = false;

				if(tid == lanid_target){
					Slot new_location_swap = location_swap | SWAP_FLAG_MASK;

					success = CompareAndSet(to_swap, location_swap, new_location_swap);
					if(success){
						location_swap = new_location_swap;

						Slot new_location_target = (location & (~MAX_KEY)) | (location_swap & MAX_KEY); 
						new_location_target &= (~EMP_FLAG_MASK);
						new_location_target &= (~CHECK_1_FLAG_MASK);
						new_location_target &= (~BITMAP_MASK);

						success = CompareAndSet(target, location, new_location_target);
						if(success){ // step 14b1a
							location = new_location_target;

							new_location_swap = (location_swap & (~SWAP_FLAG_MASK)) | EMP_FLAG_MASK | CHECK_1_FLAG_MASK ;
							/*
							if(to_swap == 7468){
								printf("Insert key: %lu, step 14b1a add CHECK_1_FLAG_MASK\n", key);
							}*/

							success = CompareAndSet(to_swap, location_swap, new_location_swap);
							if(success){ // step 14b1a1

								Slot new_location_check2 = (location_check2 & (~CHECK_2_FLAG_MASK));
								new_location_check2 &= (~(BIT<<(BITMAP_SHIFT+(to_swap_offset))));

								success = CompareAndSet(to_check_2, location_check2, new_location_check2);			
								if(success){ // step 14b1a1a
									location_check2 = new_location_check2;
									target = to_swap;
									goto_14b1a1a_step7 = true;

								} else { // step 14b1a1b
									// TODO: design fail
									printf("Insert key: %lu, step14b1a1b design fail\n", key);
								}

							} else { // step 14b1a2
								// TODO: design fail
								printf("Insert key: %lu, step14b1a2 design fail\n", key);
							}

						} else { // step 14b1b
							// TODO: design fail
							printf("Insert key: %lu, step14b1b design fail\n", key);
						}

					}

				}

				goto_14b1a1a_step7 = __shfl(goto_14b1a1a_step7, lanid_target);

				if(goto_14b1a1a_step7){
					location_check2 = __shfl(location_check2, lanid_target);
					target = __shfl(target, lanid_target);
					goto step_7;
				}

			}

			swap_list &= (~(BIT<<to_swap_offset));
		}

		// step 13b
		bool goto_13b1_step10;
			
		if(tid == 0){
			Slot new_location_check2 = (location_check2 & (~CHECK_2_FLAG_MASK));
			new_location_check2 &= (~(BIT<<(BITMAP_SHIFT+(target-to_check_2))));

			success = CompareAndSet(to_check_2, location_check2, new_location_check2);
			if(success){
				location = new_location_check2;
				location_check2 = location;
				goto_13b1_step10 = true;

			} else { // step 13b2
				// TODO: design fail
				printf("Insert key: %lu, step13b2 design fail\n", key);
			}
		}

		location_check2 = __shfl(location_check2, 0);	
		goto_13b1_step10 = __shfl(goto_13b1_step10, 0);
		if(goto_13b1_step10){
			goto step_10;
		}

	}

}

__global__ void kernel(LL* items, LL* op, LL* result)
{
/*
	for(int op_id=0;op_id<NUM_ITEMS;op_id++){
		LL itm=items[op_id];
    	result[op_id] = WRONG_POS;
    	Slot location;

    	if(op_id == 3653){
    		printf("have done %d\n",op_id-1);
    	}

    	if(op_id == 2689){
    		printf("have done %d\n",op_id-1);
    	}

    	if(op[op_id]==ADD){
      		Insert(itm, &(result[op_id])); // return 1 or 0 or WRONG_POS(need rehash)
   	 	} else if(op[op_id]==DELETE){
      		Delete(itm, &(result[op_id])); // return 1 or 0
    	} else if(op[op_id]==SEARCH){
      		Find(itm, &(result[op_id]), &location); // return slot_no or WRONG_POS
    	}
	}
*/

	for(int i=0;i<FACTOR;i++){    		// FACTOR is the number of operations per thread
    	
    	int op_id=FACTOR*blockIdx.x+i;
    	if(op_id>=NUM_ITEMS) return;

    	// Grab the operation and the associated key and execute   		
    	LL itm=items[op_id];
    	result[op_id] = WRONG_POS;
    	Slot location;

    	if(op[op_id]==ADD){
      		Insert(itm, &(result[op_id])); // return 1 or 0 or WRONG_POS(need rehash)
   	 	} else if(op[op_id]==DELETE){
      		Delete(itm, &(result[op_id])); // return 1 or 0
    	} else if(op[op_id]==SEARCH){
      		Find(itm, &(result[op_id]), &location); // return slot_no or WRONG_POS
    	}
  	}
 
}

int main(int argc, char** argv)
{

	if (argc != 3) {
    	printf("Need two arguments: percent add ops and percent delete ops (e.g., 30 50 for 30%% add and 50%% delete).\nAborting...\n");
    	exit(1);
  	}

  	int adds=atoi(argv[1]);
  	int deletes=atoi(argv[2]);

   	if (adds+deletes > 100) {
    	printf("Sum of add and delete precentages exceeds 100.\nAborting...\n");
     	exit(1);
  	}

	// Allocate hash table
	
	Slot slots[TABLE_SIZE];
	Slot * Cslots;

	int i;
	for(i=0;i<TABLE_SIZE;i++){
		slots[i] = EMP_FLAG_MASK;
	}

	#ifdef _CUTIL_H_
    	CUDA_SAFE_CALL(hipMalloc((void**)&(Cslots), sizeof(Slot)*TABLE_SIZE ));
	#else
    	hipMalloc((void**)&(Cslots), sizeof(Slot)*TABLE_SIZE );
	#endif

	#ifdef _CUTIL_H_
  		CUDA_SAFE_CALL(hipMemcpy(Cslots, slots, sizeof(Slot)*TABLE_SIZE, hipMemcpyHostToDevice));
	#else
  		hipMemcpy(Cslots, slots, sizeof(Slot)*TABLE_SIZE, hipMemcpyHostToDevice);
	#endif

	// Initialize the device memory
    init<<<1, THREADS_NUM>>>(Cslots);
  
  	LL op[NUM_ITEMS];		// Array of operations
  	LL items[NUM_ITEMS];		// Array of keys associated with operations
  	LL result[NUM_ITEMS];		// Array of outcomes
  	//LL expect_result[NUM_ITEMS]; // Array of expected result
/*
  	FILE * fp;
  	fp = fopen("/home/udms/Fanny/test/myfile_4.txt","r");
  	if(fp == NULL) exit(EXIT_FAILURE);

  	char line[100];
  	i=0;

  	while (fgets(line, 100, fp) != NULL)  {
  		char * p = strtok (line," "); 

		if(*p == 'I'){
			op[i]=ADD;
		} else if(*p == 'R'){
			op[i]=SEARCH;
		} else {
			op[i]=DELETE;
		}
		p = strtok(NULL," "); 

		if(*p == '0'){
			expect_result[i] = 0;
		} else {
			expect_result[i] = 1;
		}
		p = strtok(NULL," "); 
		
		unsigned long ul = strtoul (p, NULL, 0);
		items[i] = ul;

		i++;
	}

  	fclose(fp);
*/
  	srand(0);

  	// NUM_ITEMS is the total number of operations to execute
  	for(i=0;i<NUM_ITEMS;i++){
    	items[i]=10+rand()%KEYS;	// Keys
  	}

  	// Populate the op sequence
  	for(i=0;i<(NUM_ITEMS*adds)/100;i++){
    	op[i]=ADD;
  	}
  	for(;i<(NUM_ITEMS*(adds+deletes))/100;i++){
    	op[i]=DELETE;
  	}
  	for(;i<NUM_ITEMS;i++){
    	op[i]=SEARCH;
  	}

  	//adds=(NUM_ITEMS*adds)/100;

  	// Allocate device memory

  	LL* Citems;
  	LL* Cop;
  	LL* Cresult;

	#ifdef _CUTIL_H_
  		CUDA_SAFE_CALL(hipMalloc((void**)&Cresult, sizeof(LL)*NUM_ITEMS));
  		CUDA_SAFE_CALL(hipMalloc((void**)&Citems, sizeof(LL)*NUM_ITEMS));
  		CUDA_SAFE_CALL(hipMalloc((void**)&Cop, sizeof(LL)*NUM_ITEMS));
  		CUDA_SAFE_CALL(hipMemcpy(Citems,items, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice));
  		CUDA_SAFE_CALL(hipMemcpy(Cop, op, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice));
	#else
  		hipMalloc((void**)&Cresult, sizeof(LL)*NUM_ITEMS);
  		hipMalloc((void**)&Citems, sizeof(LL)*NUM_ITEMS);
  		hipMalloc((void**)&Cop, sizeof(LL)*NUM_ITEMS);
  		hipMemcpy(Citems,items, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice);
  		hipMemcpy(Cop, op, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice);
	#endif

	// Calculate the number of thread blocks
  	// NUM_ITEMS = total number of operations to execute
  	// NUM_THREADS = number of threads per block
  	// FACTOR = number of operations per thread

  	//int blocks=(NUM_ITEMS%FACTOR==0)?(NUM_ITEMS/FACTOR):(NUM_ITEMS/FACTOR)+1;
  	int blocks=(NUM_ITEMS%(THREADS_NUM*FACTOR)==0)?NUM_ITEMS/(THREADS_NUM*FACTOR):(NUM_ITEMS/(THREADS_NUM*FACTOR))+1;

    // Launch main kernel

  	hipEvent_t start, stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	hipEventRecord(start, 0);
  
  	kernel<<<blocks, THREADS_NUM>>>(Citems, Cop, Cresult);
  
  	hipEventRecord(stop, 0);
  	//hipEventSynchronize(start);
  	hipEventSynchronize(stop);
  	float time;
  	hipEventElapsedTime(&time, start, stop);
  	hipEventDestroy(start);
  	hipEventDestroy(stop);

  	// Print kernel execution time in milliseconds

  	printf("%lf\n",time);

  	// Check for errors

  	hipError_t error= hipGetLastError();
  	if(hipSuccess!=error){
    	printf("error:CUDA ERROR (%d) {%s}\n",error,hipGetErrorString(error));
    	exit(-1);
  	}

  	// Move results back to host memory

	#ifdef _CUTIL_H_
  		CUDA_SAFE_CALL(hipMemcpy(result, Cresult, sizeof(LL)*NUM_ITEMS, hipMemcpyDeviceToHost));
	#else
  		hipMemcpy(result, Cresult, sizeof(LL)*NUM_ITEMS, hipMemcpyDeviceToHost);
	#endif
/*
	int insert_full = 0;
	int insert_fail = 0;
	int delete_fail = 0;
	int find_fail = 0;
	for(i=0;i<NUM_ITEMS;i++){
		if(op[i]==ADD){
			if(result[i]==WRONG_POS){
				if(i == 140700){
					printf("ADD full catch, line: %d, item: %d, result: %d, expect_result: %d\n", i, (unsigned long)items[i], (int)result[i], (int)expect_result[i]);
				}
				insert_full++;
			} else if(result[i] != expect_result[i]){
				printf("ADD fail, line: %d, item: %lu, result: %d, expect_result: %d\n", i, (unsigned long)items[i], (int)result[i], (int)expect_result[i]);
				insert_fail++;
			}
   	 	} else if(op[i]==DELETE){
      		if(result[i] != expect_result[i]){
      			//printf("DELETE fail, line: %d, item: %lu, result: %d, expect_result: %d\n", i, (unsigned long)items[i], (int)result[i], (int)expect_result[i]);
				delete_fail++;
			}
    	} else if(op[i]==SEARCH){
    		if(result[i]==WRONG_POS && expect_result[i]==0){
    			;
    		} else if(result[i]!=WRONG_POS && expect_result[i]==1){
    			;
    		} else {
    			printf("SEARCH fail, line: %d, item: %lu, result: %d, expect_result: %d\n", i, (unsigned long)items[i], (int)result[i], (int)expect_result[i]);
    			find_fail++;
    		}
    	} 
	}

	printf("insert_full: %d insert_fail: %d delete_fail: %d find_fail: %d\n", insert_full, insert_fail, delete_fail, find_fail); */

	return 0;
}
